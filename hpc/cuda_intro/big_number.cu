
#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>


__global__
void add(int n, float* x, float* y) {
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;

  for (int i = index; i < n; i+= stride)
    y[i] = x[i] + y[i];
}

int main(void) {

	int N = 1<<20;
  float *x, *y;
  int blockSize = 256;
  int numBlocks = (N + blockSize - 1) / blockSize;

  // Allocate Unified Memory – accessible from CPU or GPU
  hipMallocManaged(&x, N*sizeof(float));
  hipMallocManaged(&y, N*sizeof(float));

  for (int i = 0; i < N; i++) {
    x[i] = 1.0f;
    y[i] = 2.0f;
  }

  add <<<numBlocks, blockSize>>>(N, x, y);

  // Wait for GPU to finish
  hipDeviceSynchronize();

  float maxError = 0.0f;
  for (int i = 0; i < N; i++)
    maxError = std::fmax(maxError, std::fabs(y[i] - 3.0f));

  std::cout << "Max error: " << maxError << std::endl;

  // Free memory
  hipFree(x);
  hipFree(y);

  return 0;
}
